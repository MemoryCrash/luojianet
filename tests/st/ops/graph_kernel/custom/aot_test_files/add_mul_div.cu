
#include <hip/hip_runtime.h>
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

constexpr int THREADS = 1024;

__global__ void CustomAddMulDivKernel(float *input1, float *input2, float *output1, float *output2, float *output3,
                                      size_t size) {
  auto idx = blockIdx.x * THREADS + threadIdx.x;
  if (idx < size) {
    output1[idx] = input1[idx] + input2[idx];
    output2[idx] = input1[idx] * input2[idx];
    output3[idx] = input1[idx] / input2[idx];
  }
}

extern "C" int CustomAddMulDiv(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes,
                               void *stream, void *extra) {
  hipStream_t custream = static_cast<hipStream_t>(stream);

  constexpr int OUTPUT_INDEX = 2;
  constexpr int TOTAL_PARAM_NUM = 5;

  // Users can add any check on their need. If check fails, user can return any value larger than 0 to safely exit.
  // Return value larger than 0 will cause luojianet_ms to stop computing and safely exit.
  // Specially, return 1 will show log: "Number of parameters passed is inconsistent with what the user wants".
  // return 2 will show log: "Type of parameters passed is inconsistent with what the user wants".

  // This is to check if the num of parameters the same as what the user wants.
  // There are two inputs and three outputs, so the nparam should be 5.
  if (nparam != TOTAL_PARAM_NUM) {
    return 1;
  }

  // This is to check if the type of parameters the same as what the user wants.
  for (int i = 0; i < nparam; i++) {
    if (strcmp(dtypes[i], "float32") != 0) {
      return 2;
    }
  }

  // input1's index is 0, input2's index is 1, output1's index is 2, output2's index is 3 and output3's index is 4
  void *input1 = params[0];
  void *input2 = params[1];
  void *output1 = params[2];
  void *output2 = params[3];
  void *output3 = params[4];
  size_t size = 1;

  // Cumprod of output's shape to compute elements' num
  for (int i = 0; i < ndims[OUTPUT_INDEX]; i++) {
    size *= shapes[OUTPUT_INDEX][i];
  }
  int n = size / THREADS;

  // Do the computation
  CustomAddMulDivKernel<<<n + 1, THREADS, 0, custream>>>(static_cast<float *>(input1), static_cast<float *>(input2),
                                                         static_cast<float *>(output1), static_cast<float *>(output2),
                                                         static_cast<float *>(output3), size);
  // When return 0, luojianet_ms will continue to run if this kernel could launch successfully.
  return 0;
}
